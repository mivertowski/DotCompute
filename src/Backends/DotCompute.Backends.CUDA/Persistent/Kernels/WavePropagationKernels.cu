#include "hip/hip_runtime.h"
// Copyright (c) 2025 Michael Ivertowski
// Licensed under the MIT License. See LICENSE file in the project root for license information.

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Constants for wave equation
#define WAVE_SPEED 1.0f
#define DAMPING 0.999f

// Persistent kernel for 1D acoustic wave propagation
// Uses finite difference method: u(t+1) = 2*u(t) - u(t-1) + c^2 * dt^2/dx^2 * (u[i+1] - 2*u[i] + u[i-1])
extern "C" __global__ void acoustic_wave_1d_persistent(
    float* __restrict__ u_current,    // Current time step
    float* __restrict__ u_previous,   // Previous time step  
    float* __restrict__ u_two_ago,    // Two steps ago
    int* __restrict__ control,        // Control buffer [running, iteration, error, reserved]
    const int nx,                     // Grid width
    const float dx,                   // Spatial step
    const float dt,                   // Time step
    const int max_iterations)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    const float c2_dt2_dx2 = (WAVE_SPEED * WAVE_SPEED * dt * dt) / (dx * dx);
    
    // Grid-stride loop for persistent execution
    while (control[0] == 1 && control[1] < max_iterations) {
        // Process all points using grid-stride loop
        for (int i = tid; i < nx; i += stride) {
            // Skip boundary points (Dirichlet boundary conditions)
            if (i == 0 || i == nx - 1) {
                u_current[i] = 0.0f;
                continue;
            }
            
            // Finite difference wave equation
            float laplacian = u_previous[i + 1] - 2.0f * u_previous[i] + u_previous[i - 1];
            u_current[i] = 2.0f * u_previous[i] - u_two_ago[i] + c2_dt2_dx2 * laplacian;
            u_current[i] *= DAMPING; // Apply damping
        }
        
        // Synchronize all threads before swapping buffers
        __syncthreads();
        
        // Only thread 0 updates control and swaps pointers
        if (tid == 0) {
            // Rotate buffers: two_ago <- previous <- current <- two_ago
            float* temp = u_two_ago;
            u_two_ago = u_previous;
            u_previous = u_current;
            u_current = temp;
            
            // Update iteration counter
            atomicAdd(&control[1], 1);
        }
        
        __syncthreads();
    }
}

// Persistent kernel for 2D acoustic wave propagation
extern "C" __global__ void acoustic_wave_2d_persistent(
    float* __restrict__ u_current,
    float* __restrict__ u_previous,
    float* __restrict__ u_two_ago,
    int* __restrict__ control,
    const int nx,
    const int ny,
    const float dx,
    const float dy,
    const float dt,
    const int max_iterations)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total_points = nx * ny;
    
    const float c2_dt2_dx2 = (WAVE_SPEED * WAVE_SPEED * dt * dt) / (dx * dx);
    const float c2_dt2_dy2 = (WAVE_SPEED * WAVE_SPEED * dt * dt) / (dy * dy);
    
    while (control[0] == 1 && control[1] < max_iterations) {
        for (int idx = tid; idx < total_points; idx += stride) {
            const int i = idx % nx;
            const int j = idx / nx;
            
            // Skip boundary points
            if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1) {
                u_current[idx] = 0.0f;
                continue;
            }
            
            // 2D Laplacian
            float laplacian_x = u_previous[idx + 1] - 2.0f * u_previous[idx] + u_previous[idx - 1];
            float laplacian_y = u_previous[idx + nx] - 2.0f * u_previous[idx] + u_previous[idx - nx];
            
            u_current[idx] = 2.0f * u_previous[idx] - u_two_ago[idx] 
                           + c2_dt2_dx2 * laplacian_x 
                           + c2_dt2_dy2 * laplacian_y;
            u_current[idx] *= DAMPING;
        }
        
        __syncthreads();
        
        if (tid == 0) {
            float* temp = u_two_ago;
            u_two_ago = u_previous;
            u_previous = u_current;
            u_current = temp;
            atomicAdd(&control[1], 1);
        }
        
        __syncthreads();
    }
}

// Persistent kernel for 3D acoustic wave propagation
extern "C" __global__ void acoustic_wave_3d_persistent(
    float* __restrict__ u_current,
    float* __restrict__ u_previous,
    float* __restrict__ u_two_ago,
    int* __restrict__ control,
    const int nx,
    const int ny,
    const int nz,
    const float dx,
    const float dy,
    const float dz,
    const float dt,
    const int max_iterations)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total_points = nx * ny * nz;
    
    const float c2_dt2_dx2 = (WAVE_SPEED * WAVE_SPEED * dt * dt) / (dx * dx);
    const float c2_dt2_dy2 = (WAVE_SPEED * WAVE_SPEED * dt * dt) / (dy * dy);
    const float c2_dt2_dz2 = (WAVE_SPEED * WAVE_SPEED * dt * dt) / (dz * dz);
    
    while (control[0] == 1 && control[1] < max_iterations) {
        for (int idx = tid; idx < total_points; idx += stride) {
            const int i = idx % nx;
            const int j = (idx / nx) % ny;
            const int k = idx / (nx * ny);
            
            // Skip boundary points
            if (i == 0 || i == nx - 1 || 
                j == 0 || j == ny - 1 || 
                k == 0 || k == nz - 1) {
                u_current[idx] = 0.0f;
                continue;
            }
            
            // 3D Laplacian
            float laplacian_x = u_previous[idx + 1] - 2.0f * u_previous[idx] + u_previous[idx - 1];
            float laplacian_y = u_previous[idx + nx] - 2.0f * u_previous[idx] + u_previous[idx - nx];
            float laplacian_z = u_previous[idx + nx*ny] - 2.0f * u_previous[idx] + u_previous[idx - nx*ny];
            
            u_current[idx] = 2.0f * u_previous[idx] - u_two_ago[idx]
                           + c2_dt2_dx2 * laplacian_x
                           + c2_dt2_dy2 * laplacian_y
                           + c2_dt2_dz2 * laplacian_z;
            u_current[idx] *= DAMPING;
        }
        
        __syncthreads();
        
        if (tid == 0) {
            float* temp = u_two_ago;
            u_two_ago = u_previous;
            u_previous = u_current;
            u_current = temp;
            atomicAdd(&control[1], 1);
        }
        
        __syncthreads();
    }
}

// Cooperative groups version for grid-wide synchronization
extern "C" __global__ void acoustic_wave_2d_cooperative(
    float* __restrict__ u_current,
    float* __restrict__ u_previous,
    float* __restrict__ u_two_ago,
    int* __restrict__ control,
    const int nx,
    const int ny,
    const float dx,
    const float dy,
    const float dt,
    const int max_iterations)
{
    cg::grid_group grid = cg::this_grid();
    const int tid = grid.thread_rank();
    const int stride = grid.size();
    const int total_points = nx * ny;
    
    const float c2_dt2_dx2 = (WAVE_SPEED * WAVE_SPEED * dt * dt) / (dx * dx);
    const float c2_dt2_dy2 = (WAVE_SPEED * WAVE_SPEED * dt * dt) / (dy * dy);
    
    while (control[0] == 1 && control[1] < max_iterations) {
        for (int idx = tid; idx < total_points; idx += stride) {
            const int i = idx % nx;
            const int j = idx / nx;
            
            if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1) {
                u_current[idx] = 0.0f;
                continue;
            }
            
            float laplacian_x = u_previous[idx + 1] - 2.0f * u_previous[idx] + u_previous[idx - 1];
            float laplacian_y = u_previous[idx + nx] - 2.0f * u_previous[idx] + u_previous[idx - nx];
            
            u_current[idx] = 2.0f * u_previous[idx] - u_two_ago[idx]
                           + c2_dt2_dx2 * laplacian_x
                           + c2_dt2_dy2 * laplacian_y;
            u_current[idx] *= DAMPING;
        }
        
        // Grid-wide synchronization
        grid.sync();
        
        if (tid == 0) {
            float* temp = u_two_ago;
            u_two_ago = u_previous;
            u_previous = u_current;
            u_current = temp;
            atomicAdd(&control[1], 1);
        }
        
        grid.sync();
    }
}

// Helper kernel to initialize wave field with a Gaussian pulse
extern "C" __global__ void initialize_gaussian_pulse_2d(
    float* field,
    const int nx,
    const int ny,
    const float center_x,
    const float center_y,
    const float sigma,
    const float amplitude)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int total_points = nx * ny;
    
    for (int idx = tid; idx < total_points; idx += stride) {
        const int i = idx % nx;
        const int j = idx / nx;
        
        const float x = (float)i - center_x;
        const float y = (float)j - center_y;
        const float r2 = x * x + y * y;
        
        field[idx] = amplitude * expf(-r2 / (2.0f * sigma * sigma));
    }
}

// Helper kernel to add source term
extern "C" __global__ void add_source_term(
    float* field,
    const int source_idx,
    const float amplitude,
    const float frequency,
    const float time)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        field[source_idx] += amplitude * sinf(2.0f * 3.14159265f * frequency * time);
    }
}